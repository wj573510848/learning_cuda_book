
#include <hip/hip_runtime.h>
# include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU!\n");
}

int main(void)
{
    hello_from_gpu<<<2,4>>>(); // <<<>>>里面指代 线程块数,每个线程块中的线程数
    hipDeviceSynchronize();
    return 0;
}