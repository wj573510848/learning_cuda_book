
#include <hip/hip_runtime.h>
# include <stdio.h>

__global__ void hello_from_gpu()
{   
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("Hello World from block %d and thread %d!\n",bid,tid);
}

int main(void)
{
    hello_from_gpu<<<2,4>>>(); // <<<>>>里面指代 线程块数,每个线程块中的线程数
    hipDeviceSynchronize();
    return 0;
}